#include "hip/hip_runtime.h"
#include "../include/host_code.cuh"
#include "../include/kernels.cuh"
#include <hipcub/hipcub.hpp>
#include <numeric>

using std::vector;

/**
 * @brief Perform final reduction of a pair of masks on cpu.
 *
 * @param results Partially reduced results to be reduced to a single value.
 * @return MinResult The final pixel pair of minimum distance.
 */
MinResult cpu_reduction(PinnedVector<MinResult> results) {
  MinResult min_result{INT_MAX, -1, -1};
  for (MinResult &result : results) {
    if (result.distance < min_result.distance)
      min_result = result;
  }
  return min_result;
}

/**
 * @brief Sort non-zero values and corresponding indices by value.
 *
 * Uses CUB's RadixSort to sort values and indices using values as key. The
 * purpose of the sorting is not to order the values in any particular order,
 * but to lump all indices corresponding to each mask together.
 *
 * The function reinterpret casts the ints to unsigned because they are positive
 * anyway, and radix sort is faster for unsigned ints. Besides, we do not care
 * about the order, we just want identical values to be adjacent.
 *
 * To increase sorting efficiency, only the 8 least significant bits are used in
 * the sort. This means that masks with values greater than 256 will fail to be
 * sorted.
 *
 * The sorted results are stored in the input arrays.
 *
 * @param d_mask_idxs The unsorted indices of non-zero values, in gpu memory.
 * @param d_mask_values The corresponding indices, in gpu memory.
 * @param h_num_mask_pixels The number of non-zero elements, in cpu memory.
 */
void sort_nonzeros(int *&d_mask_idxs, int *&d_mask_values,
                   int h_num_mask_pixels) {
  int *d_idxs_buffer, *d_values_buffer;
  CUDA_CHECK(hipMalloc(&d_idxs_buffer, sizeof(int) * h_num_mask_pixels));
  CUDA_CHECK(hipMalloc(&d_values_buffer, sizeof(int) * h_num_mask_pixels));

  // Create a DoubleBuffer to wrap the pair of device pointers
  hipcub::DoubleBuffer<unsigned int> d_keys(
      reinterpret_cast<unsigned int *>(d_mask_values),
      reinterpret_cast<unsigned int *>(d_values_buffer));

  hipcub::DoubleBuffer<unsigned int> d_values(
      reinterpret_cast<unsigned int *>(d_mask_idxs),
      reinterpret_cast<unsigned int *>(d_idxs_buffer));

  // Determine temporary device storage requirements
  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                             d_keys, d_values,
                                             h_num_mask_pixels, 0, 8));

  // Allocate temporary storage
  CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));

  // Run sorting operation
  CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                             d_keys, d_values,
                                             h_num_mask_pixels, 0, 8));
  CUDA_CHECK(hipFree(d_temp_storage));
  bool output_in_buffer =
      (reinterpret_cast<int *>(d_keys.Current()) == d_values_buffer);
  if (output_in_buffer) {
    std::swap(d_mask_idxs, d_idxs_buffer);
    std::swap(d_mask_values, d_values_buffer);
  }
  CUDA_CHECK(hipFree(d_idxs_buffer));
  CUDA_CHECK(hipFree(d_values_buffer));
}

/**
 * @brief Find the number of masks, and their values and sizes using CUB.
 *
 * @param d_sorted_mask_values A sorted array of all mask values.
 * @param num_mask_pixels The number of non-zero values.
 * @return std::tuple<vector<int>, vector<int>, int> Number of masks, values,
 * and sizes.
 */
std::tuple<vector<int>, vector<int>, int> encode_runs(int *d_sorted_mask_values,
                                                      int num_mask_pixels) {
  int *d_unique_values;
  CUDA_CHECK(hipMalloc(&d_unique_values, sizeof(int) * 256));
  int *d_mask_sizes;
  CUDA_CHECK(hipMalloc(&d_mask_sizes, sizeof(int) * 256));
  int *d_num_unique_values;
  CUDA_CHECK(hipMalloc(&d_num_unique_values, sizeof(int)));

  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  CUDA_CHECK(hipcub::DeviceRunLengthEncode::Encode(
      d_temp_storage, temp_storage_bytes, d_sorted_mask_values, d_unique_values,
      d_mask_sizes, d_num_unique_values, num_mask_pixels));

  CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));

  CUDA_CHECK(hipcub::DeviceRunLengthEncode::Encode(
      d_temp_storage, temp_storage_bytes, d_sorted_mask_values, d_unique_values,
      d_mask_sizes, d_num_unique_values, num_mask_pixels));

  CUDA_CHECK(hipFree(d_temp_storage));

  int h_num_unique_values;
  CUDA_CHECK(hipMemcpy(&h_num_unique_values, d_num_unique_values, sizeof(int),
                        hipMemcpyDeviceToHost));

  vector<int> h_unique_values(h_num_unique_values);
  vector<int> h_mask_sizes(h_num_unique_values);

  CUDA_CHECK(hipMemcpy(h_unique_values.data(), d_unique_values,
                        sizeof(int) * h_num_unique_values,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(h_mask_sizes.data(), d_mask_sizes,
                        sizeof(int) * h_num_unique_values,
                        hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(d_unique_values));
  CUDA_CHECK(hipFree(d_num_unique_values));
  CUDA_CHECK(hipFree(d_mask_sizes));

  return std::make_tuple(h_unique_values, h_mask_sizes, h_num_unique_values);
}

/**
 * @brief Take in an image, find mask edges, and format them for distance
 * calculations.
 *
 * First, a handwritten kernel is used to find all mask edges, and put their
 * one-dimensional indices as well as values in arrays. Then, the two arrays are
 * sorted by mask value. Finally, the sorted values are analyzed to get the mask
 * values, sizes, and number of masks.
 *
 * The function returns the mask values, sizes, and number of masks in a tuple.
 * The mask indices and values are stored in the input arrays in gpu memory.
 *
 * @param h_image The input image, in cpu memory.
 * @param img_height The image height in pixels.
 * @param img_width The image width in pixels.
 * @param d_mask_idxs Array in gpu memory of size img_height * img_width.
 * @param d_mask_values Array in gpu memory of size img_height * img_width.
 * @return std::tuple<vector<int>, vector<int>, int> Mask values, sizes, and
 * number of masks.
 */
std::tuple<vector<int>, vector<int>, int>
index_masks(const int *const h_image, int img_height, int img_width,
            int *&d_mask_idxs, int *&d_mask_values) {
  int total_pixels = img_height * img_width;

  int *d_image;
  CUDA_CHECK(hipMalloc(&d_image, sizeof(int) * total_pixels));
  CUDA_CHECK(hipMemcpy(d_image, h_image, sizeof(int) * total_pixels,
                        hipMemcpyHostToDevice));

  // Find & index edges
  dim3 block_dims = get_block_dims_indexing();
  dim3 grid_dims = get_grid_dims_indexing(img_height, img_width);
  int *d_num_nonzeros;
  CUDA_CHECK(hipMalloc(&d_num_nonzeros, sizeof(int)));
  CUDA_CHECK(hipMemset(d_num_nonzeros, 0, sizeof(int)));
  index_edges<<<grid_dims, block_dims>>>(d_image, img_height, img_width,
                                         d_mask_idxs, d_mask_values,
                                         d_num_nonzeros);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipFree(d_image));

  int h_num_nonzeros;
  CUDA_CHECK(hipMemcpy(&h_num_nonzeros, d_num_nonzeros, sizeof(int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(d_num_nonzeros));

  // Sort
  sort_nonzeros(d_mask_idxs, d_mask_values, h_num_nonzeros);

  // Encode runs
  return encode_runs(d_mask_values, h_num_nonzeros);
}

/**
 * @brief Launch the min_pair_thread_per_a kernel and final reduction kernel.
 *
 * Because the kernel parallelizes over a's, the a and b arrays are swapped if
 * there are more b's than a's to allow for maximum parallelization.
 *
 * @param num_as Number of as.
 * @param num_bs Number of bs.
 * @param img_width Image width in pixels.
 * @param d_as A indices in gpu memory.
 * @param d_bs B indices in gpu memory.
 * @param h_result Variable where result will be stored.
 * @param stream Cuda stream to launch kernel in.
 */
void launch_min_pair_thread_per_a(int num_as, int num_bs, const int img_width,
                                  int *d_as, int *d_bs, MinResult &h_result,
                                  hipStream_t stream) {

  // Kernel parallelizes over a's. Launch kernel with the larger mask as a.
  bool swapped = num_as < num_bs;
  if (swapped) {
    std::swap(num_as, num_bs);
    std::swap(d_as, d_bs);
  }

  int num_blocks = get_grid_size_distance(num_as);
  int block_size = get_block_size_distance();
  MinResult *d_results;
  CUDA_CHECK(
      hipMallocAsync(&d_results, sizeof(MinResult) * num_blocks, stream));

  min_distances_thread_per_a<<<num_blocks, block_size, 0, stream>>>(
      d_as, d_bs, num_as, num_bs, img_width, d_results, swapped);
  CUDA_CHECK(hipGetLastError());

  final_reduction<<<1, block_size, 0, stream>>>(d_results, num_blocks,
                                                d_results);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipMemcpyAsync(&h_result, d_results, sizeof(MinResult),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipFreeAsync(d_results, stream));
}

/**
 * @brief Launch the min_pair_thread_per_pair kernel and final reduction kernel.
 *
 * Because the kernel expects the indices as 2d indices stored as int2's, a
 * thrust transformation is first used to transform the 1d indices to 2d.
 *
 * @param num_as Number of a pixels.
 * @param num_bs Number of b pixels.
 * @param img_width Image width in pixels.
 * @param d_as A indices in gpu memory.
 * @param d_bs B indices in gpu memory.
 * @param h_result Variable where result will be stored.
 * @param stream Cuda stream to launch kernel in.
 */
void launch_min_pair_thread_per_pair(const int num_as, const int num_bs,
                                     const int img_width, int *d_as, int *d_bs,
                                     MinResult &h_results,
                                     hipStream_t stream) {

  dim3 block_dims = get_block_dims_distance_2d();
  dim3 grid_dims = get_grid_dims_2d(num_as, num_bs);
  int num_blocks = (int)grid_dims.x * (int)grid_dims.y;

  MinResult *d_results;
  CUDA_CHECK(
      hipMallocAsync(&d_results, sizeof(MinResult) * num_blocks, stream));

  int2 *d_points_a, *d_points_b;
  CUDA_CHECK(hipMallocAsync(&d_points_a, sizeof(int2) * num_as, stream));
  CUDA_CHECK(hipMallocAsync(&d_points_b, sizeof(int2) * num_bs, stream));
  make_points(d_as, d_bs, num_as, num_bs, img_width, d_points_a, d_points_b,
              stream);

  min_distances_thread_per_pair<<<grid_dims, block_dims, 0, stream>>>(
      d_points_a, d_points_b, num_as, num_bs, img_width, d_results);
  CUDA_CHECK(hipGetLastError());

  int block_size = get_block_size_distance();
  final_reduction<<<1, block_size, 0, stream>>>(d_results, num_blocks,
                                                d_results);
  CUDA_CHECK(hipGetLastError());

  CUDA_CHECK(hipMemcpyAsync(&h_results, d_results, sizeof(MinResult),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipFreeAsync(d_points_a, stream));
  CUDA_CHECK(hipFreeAsync(d_points_b, stream));
  CUDA_CHECK(hipFreeAsync(d_results, stream));
}

/**
 * @brief Get the pair of pixels in a and b that are closest to each other.
 *
 * This function will delegate the work to the kernel with one thread per a or
 * the kernel with one thread per pixel pair based on the number of masks,
 * maximum mask size, and total number of masked pixels.
 *
 * @param d_as A indices in gpu memory.
 * @param d_bs B indices in gpu memory.
 * @param num_as Number of a pixels.
 * @param num_bs Number of b pixels.
 * @param img_width Image width in pixels.
 * @param h_result Variable where result will be stored.
 * @param stream Cuda stream to launch kernel in.
 * @param use_kernel_1 Whether to use kernel 1 or 2.
 */
void launch_min_pair_kernel(int *d_as, int *d_bs, int num_as, int num_bs,
                            int img_width, MinResult &h_result,
                            hipStream_t stream, bool use_kernel_1) {
  if (use_kernel_1) {
    launch_min_pair_thread_per_a(num_as, num_bs, img_width, d_as, d_bs,
                                 h_result, stream);
  } else {
    launch_min_pair_thread_per_pair(num_as, num_bs, img_width, d_as, d_bs,
                                    h_result, stream);
  }
}

/**
 * @brief For each combination of masks, launch find the pixels with minimum
 * distance.
 *
 * This function launches each kernel in its own separate cuda stream.
 *
 * @param mask_values The values of all masks.
 * @param mask_sizes The sizes of all masks.
 * @param num_masks The total number of masks.
 * @param d_sorted_idxs 1d mask indices sorted by mask value, in gpu memory.
 * @param img_width Image width in pixels.
 * @return vector<PinnedVector<MinResult>> 2d array of min pixel pair between
 * all masks.
 */
vector<PinnedVector<MinResult>>
pair_reductions(vector<int> mask_values, vector<int> mask_sizes, int num_masks,
                int *d_sorted_idxs, int img_width) {

  // Kernel 2 is only faster if there are very few pixel combinations in total
  // to check
  int num_nonzeros = std::reduce(mask_sizes.begin(), mask_sizes.end());
  int max_mask_size = *std::max_element(mask_sizes.begin(), mask_sizes.end());
  int num_mask_combinations = (num_masks * (num_masks - 1)) / 2;
  bool use_kernel_1 =
      !(num_mask_combinations < 6 && max_mask_size < 75 && num_nonzeros < 150);

  int num_streams = std::min(num_mask_combinations, 32);
  vector<hipStream_t> streams(num_streams);
  for (auto &stream : streams) {
    CUDA_CHECK(hipStreamCreate(&stream));
  }

  vector<PinnedVector<MinResult>> h_results(num_masks,
                                            PinnedVector<MinResult>(num_masks));

  int stream_number = 0;
  int a_offset = 0;
  for (int i = 0; i < num_masks; ++i) {
    int a = mask_values[i];
    int num_as = mask_sizes[i];
    int *a_ptr = d_sorted_idxs + a_offset;

    int b_offset = a_offset + num_as;
    for (int j = i + 1; j < num_masks; ++j) {
      int num_bs = mask_sizes[j];
      int b = mask_values[j];
      int *b_ptr = d_sorted_idxs + b_offset;

      launch_min_pair_kernel(
          a_ptr, b_ptr, num_as, num_bs, img_width, h_results[i][j],
          streams[stream_number % num_streams], use_kernel_1);

      b_offset += num_bs;
      ++stream_number;
    }
    a_offset += num_as;
  }

  for (auto &stream : streams) {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  return h_results;
}

/**
 * @brief For all masks in image, get the pixels with minimum distance between
 * two masks.
 *
 * This function serves as the entry point of the entire calculation. It first
 * launches kernels for finding and indexing mask edges, and then launches
 * distance reduction kernels for each mask combination. Finally, it formats the
 * results to a 2d array of Pair objects, with the diagonal elements being dummy
 * values.
 *
 * @param h_image Image array in cpu memory.
 * @param img_height Image height in pixels
 * @param img_width Image width in pixels.
 * @return vector<vector<Pair>> All pixels pairs with minimum distance between
 * masks.
 */
vector<vector<Pair>> get_pairs(const int *const h_image, const int img_height,
                               const int img_width) {
  int total_pixels = img_width * img_height;

  int *d_sorted_idxs;
  int *d_sorted_values;
  CUDA_CHECK(hipMalloc(&d_sorted_idxs, sizeof(int) * total_pixels));
  CUDA_CHECK(hipMalloc(&d_sorted_values, sizeof(int) * total_pixels));

  // Index all masks
  auto [unique_values, mask_sizes, num_unique_values] = index_masks(
      h_image, img_height, img_width, d_sorted_idxs, d_sorted_values);
  CUDA_CHECK(hipFree(d_sorted_values));

  // Calculate (not completely reduced) pixel pairings between all masks
  vector<PinnedVector<MinResult>> h_pixel_pairs = pair_reductions(
      unique_values, mask_sizes, num_unique_values, d_sorted_idxs, img_width);
  CUDA_CHECK(hipFree(d_sorted_idxs));

  vector<vector<Pair>> pairs(num_unique_values,
                             vector<Pair>(num_unique_values));
  for (int i = 0; i < num_unique_values; ++i) {
    for (int j = i; j < num_unique_values; ++j) {
      MinResult result;
      if (i == j)
        result = MinResult{0, -1, -1};
      else
        result = h_pixel_pairs[i][j];
      Pair pair;
      pair.a = unique_values[i];
      pair.ax = result.a_idx % img_width;
      pair.ay = result.a_idx / img_width;
      pair.b = unique_values[j];
      pair.bx = result.b_idx % img_width;
      pair.by = result.b_idx / img_width;
      pair.distance = sqrt(result.distance);

      pairs[i][j] = pair;
      pairs[j][i] = pair.transpose();
    }
  }

  return pairs;
}
