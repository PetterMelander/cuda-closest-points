#include "hip/hip_runtime.h"
#include "../include/host_code.cuh"
#include "../include/kernels.cuh"
#include <climits>
#include <cstddef>
#include <hipcub/hipcub.hpp>
#include <tuple>
#include <utility>
#include <vector>

using std::vector;

MinResult cpu_reduction(vector<MinResult> results) {
  MinResult min_result{INT_MAX, -1, -1};
  for (MinResult &result : results) {
    if (result.distance < min_result.distance)
      min_result = result;
  }
  return min_result;
}

void sort_nonzeros(int *&d_nonzero_idxs, int *&d_nonzero_values,
                   int h_num_nonzeros) {
  // This code reinterpret casts the ints to unsigned because they are positive
  // anyway, and radix sort is faster for unsigned ints. Besides, we do not care
  // about the order, we just want identical values to be adjacent.

  int *d_idxs_buffer, *d_values_buffer;
  CUDA_CHECK(hipMalloc(&d_idxs_buffer, sizeof(int) * h_num_nonzeros));
  CUDA_CHECK(hipMalloc(&d_values_buffer, sizeof(int) * h_num_nonzeros));

  // Create a DoubleBuffer to wrap the pair of device pointers
  hipcub::DoubleBuffer<unsigned int> d_keys(
      reinterpret_cast<unsigned int *>(d_nonzero_values),
      reinterpret_cast<unsigned int *>(d_values_buffer));

  hipcub::DoubleBuffer<unsigned int> d_values(
      reinterpret_cast<unsigned int *>(d_nonzero_idxs),
      reinterpret_cast<unsigned int *>(d_idxs_buffer));

  // Determine temporary device storage requirements
  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                             d_keys, d_values, h_num_nonzeros,
                                             0, 8));

  // Allocate temporary storage
  CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));

  // Run sorting operation
  CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                             d_keys, d_values, h_num_nonzeros,
                                             0, 8));
  CUDA_CHECK(hipFree(d_temp_storage));
  bool output_in_buffer =
      (reinterpret_cast<int *>(d_keys.Current()) == d_values_buffer);
  if (output_in_buffer) {
    std::swap(d_idxs_buffer, d_nonzero_idxs);
    std::swap(d_values_buffer, d_nonzero_values);
  }
  CUDA_CHECK(hipFree(d_idxs_buffer));
  CUDA_CHECK(hipFree(d_values_buffer));
}

std::tuple<vector<int>, vector<int>, int>
encode_runs(int *d_sorted_nonzero_values, int num_nonzeros) {
  int *d_unique_values;
  CUDA_CHECK(hipMalloc(&d_unique_values, sizeof(int) * 256));
  int *d_mask_sizes;
  CUDA_CHECK(hipMalloc(&d_mask_sizes, sizeof(int) * 256));
  int *d_num_unique_values;
  CUDA_CHECK(hipMalloc(&d_num_unique_values, sizeof(int)));

  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  CUDA_CHECK(hipcub::DeviceRunLengthEncode::Encode(
      d_temp_storage, temp_storage_bytes, d_sorted_nonzero_values,
      d_unique_values, d_mask_sizes, d_num_unique_values, num_nonzeros));

  CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));

  CUDA_CHECK(hipcub::DeviceRunLengthEncode::Encode(
      d_temp_storage, temp_storage_bytes, d_sorted_nonzero_values,
      d_unique_values, d_mask_sizes, d_num_unique_values, num_nonzeros));

  CUDA_CHECK(hipFree(d_temp_storage));

  int h_num_unique_values;
  CUDA_CHECK(hipMemcpy(&h_num_unique_values, d_num_unique_values, sizeof(int),
                        hipMemcpyDeviceToHost));

  vector<int> h_unique_values(h_num_unique_values);
  vector<int> h_mask_sizes(h_num_unique_values);

  CUDA_CHECK(hipMemcpy(h_unique_values.data(), d_unique_values,
                        sizeof(int) * h_num_unique_values,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(h_mask_sizes.data(), d_mask_sizes,
                        sizeof(int) * h_num_unique_values,
                        hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(d_unique_values));
  CUDA_CHECK(hipFree(d_num_unique_values));
  CUDA_CHECK(hipFree(d_mask_sizes));

  return std::make_tuple(h_unique_values, h_mask_sizes, h_num_unique_values);
}

std::tuple<vector<int>, vector<int>, int>
index_masks(const int *const h_image, int total_pixels, int *&d_nonzero_idxs,
               int *&d_nonzero_values) {

  int *d_image;
  CUDA_CHECK(hipMalloc(&d_image, sizeof(int) * total_pixels));
  CUDA_CHECK(hipMemcpy(d_image, h_image, sizeof(int) * total_pixels,
                        hipMemcpyHostToDevice));

  // Index shapes
  int *d_num_nonzeros;
  CUDA_CHECK(hipMalloc(&d_num_nonzeros, sizeof(int)));

  int num_blocks =
      num_blocks_max_occupancy(find_nonzeros, THREADS_PER_BLOCK,
                               sizeof(int) * 2 * TILE_SIZE_INDEXING, 1.5f);
  find_nonzeros<<<num_blocks, THREADS_PER_BLOCK>>>(
      d_image, total_pixels, d_nonzero_idxs, d_nonzero_values, d_num_nonzeros);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipFree(d_image));

  int h_num_nonzeros;
  CUDA_CHECK(hipMemcpy(&h_num_nonzeros, d_num_nonzeros, sizeof(int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(d_num_nonzeros));

  // Sort
  sort_nonzeros(d_nonzero_idxs, d_nonzero_values, h_num_nonzeros);

  // Encode runs
  return encode_runs(d_nonzero_values, h_num_nonzeros);
}

vector<MinResult> launch_min_pair_thread_per_a(int num_as, int num_bs,
                                               const int img_width, int *d_as,
                                               int *d_bs) {

  // Kernel parallelizes over a's. Launch kernel with the larger mask as a.
  bool swapped = num_as < num_bs;
  if (swapped) {
    std::swap(num_as, num_bs);
    std::swap(d_as, d_bs);
  }

  int num_blocks = (num_as + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  MinResult *d_results;
  CUDA_CHECK(hipMalloc(&d_results, sizeof(MinResult) * num_blocks));

  min_distances_thread_per_a<<<num_blocks, THREADS_PER_BLOCK>>>(
      d_as, d_bs, num_as, num_bs, img_width, d_results);
  CUDA_CHECK(hipGetLastError());

  // Copy results to vector on host
  vector<MinResult> h_results(num_blocks);
  CUDA_CHECK(hipMemcpy(h_results.data(), d_results,
                        sizeof(MinResult) * num_blocks,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(d_results));

  // Swap back
  if (swapped) {
    for (MinResult &result : h_results) {
      std::swap(result.a_idx, result.b_idx);
    }
  }
  return h_results;
}

vector<MinResult> launch_min_pair_thread_per_pair(const int num_as,
                                                  const int num_bs,
                                                  const int img_width,
                                                  int *d_as, int *d_bs) {

  uint block_dim = 16;
  dim3 block_size{block_dim, block_dim};
  int num_blocks = num_blocks_max_occupancy(
      min_distances_thread_per_pair, block_dim * block_dim,
      sizeof(MinResult) * WARP_SIZE, 1.42f);
  uint grid_dim = (uint)sqrt(num_blocks);
  dim3 grid_size{grid_dim, grid_dim};
  num_blocks = (int)grid_dim * (int)grid_dim;

  MinResult *d_results;
  CUDA_CHECK(hipMalloc(&d_results, sizeof(MinResult) * num_blocks));

  int2 *d_points_a, *d_points_b;
  CUDA_CHECK(hipMalloc(&d_points_a, sizeof(int2) * num_as));
  CUDA_CHECK(hipMalloc(&d_points_b, sizeof(int2) * num_bs));
  make_points(d_as, d_bs, num_as, num_bs, img_width, d_points_a, d_points_b);

  min_distances_thread_per_pair<<<grid_size, block_size>>>(
      d_points_a, d_points_b, num_as, num_bs, img_width, d_results);
  CUDA_CHECK(hipGetLastError());

  // Copy results to vector on host
  vector<MinResult> h_results(num_blocks);
  CUDA_CHECK(hipMemcpy(h_results.data(), d_results,
                        sizeof(MinResult) * num_blocks,
                        hipMemcpyDeviceToHost));

  // Unlike the other min_distance kernel whis one returns MinResults with
  // indices indicating at which indices in d_points the coordinates are
  // located. Before returning, get the actual linear indices.
  vector<int2> h_points_a(num_as);
  CUDA_CHECK(hipMemcpy(h_points_a.data(), d_points_a, sizeof(int2) * num_as,
                        hipMemcpyDeviceToHost));
  vector<int2> h_points_b(num_bs);
  CUDA_CHECK(hipMemcpy(h_points_b.data(), d_points_b, sizeof(int2) * num_bs,
                        hipMemcpyDeviceToHost));

  for (MinResult &result : h_results) {
    int2 a = h_points_a[result.a_idx];
    int2 b = h_points_b[result.b_idx];
    result.a_idx = a.x * img_width + a.y;
    result.b_idx = b.x * img_width + b.y;
  }

  CUDA_CHECK(hipFree(d_points_a));
  CUDA_CHECK(hipFree(d_points_b));
  CUDA_CHECK(hipFree(d_results));

  return h_results;
}

vector<MinResult> get_min_pairs(int *d_as, int *d_bs, int num_as, int num_bs,
                                int img_width) {
  long long num_pairs = (long long)num_as * (long long)num_bs;
  int max_mask_size = std::max(num_as, num_bs);
  if (num_pairs > (long long)INT_MAX || max_mask_size > 5000) {
    return launch_min_pair_thread_per_a(num_as, num_bs, img_width, d_as, d_bs);
  } else {
    return launch_min_pair_thread_per_pair(num_as, num_bs, img_width, d_as,
                                           d_bs);
  }
}

vector<vector<vector<MinResult>>>
initial_pair_reductions(vector<int> unique_values, vector<int> mask_sizes,
                        int num_unique_values, int *d_sorted_idxs,
                        int img_width) {
  vector<vector<vector<MinResult>>> h_unreduced_pixel_pairs(
      num_unique_values, vector<vector<MinResult>>(num_unique_values));
  int a_offset = 0;
  for (int i = 0; i < num_unique_values; ++i) {
    int a = unique_values[i];
    int num_as = mask_sizes[i];
    int *a_ptr = d_sorted_idxs + a_offset;

    int b_offset = a_offset + num_as;
    for (int j = i + 1; j < num_unique_values; ++j) {
      int num_bs = mask_sizes[j];
      int b = unique_values[j];
      int *b_ptr = d_sorted_idxs + b_offset;

      vector<MinResult> h_result_vector =
          get_min_pairs(a_ptr, b_ptr, num_as, num_bs, img_width);

      h_unreduced_pixel_pairs[i][j] = h_result_vector;
      h_unreduced_pixel_pairs[j][i] = h_result_vector;
      b_offset += num_bs;
    }
    a_offset += num_as;
  }
  return h_unreduced_pixel_pairs;
}

vector<vector<Pair>> get_pairs(const int *const h_image, const int img_width,
                               const int img_height) {
  int total_pixels = img_width * img_height;

  int *d_sorted_idxs;
  int *d_sorted_values;
  CUDA_CHECK(hipMalloc(&d_sorted_idxs, sizeof(int) * total_pixels));
  CUDA_CHECK(hipMalloc(&d_sorted_values, sizeof(int) * total_pixels));

  // Index all masks
  auto [unique_values, mask_sizes, num_unique_values] =
      index_masks(h_image, total_pixels, d_sorted_idxs, d_sorted_values);
  CUDA_CHECK(hipFree(d_sorted_values));

  // Calculate (not completely reduced) pixel pairings between all masks
  vector<vector<vector<MinResult>>> h_unreduced_pixel_pairs =
      initial_pair_reductions(unique_values, mask_sizes, num_unique_values,
                              d_sorted_idxs, img_width);
  CUDA_CHECK(hipFree(d_sorted_idxs));

  // For each mask pair, do final reduction on cpu
  vector<vector<Pair>> pairs(num_unique_values,
                             vector<Pair>(num_unique_values));
  for (int i = 0; i < num_unique_values; ++i) {
    for (int j = i; j < num_unique_values; ++j) {
      MinResult result;
      if (i == j)
        result = MinResult{0, -1, -1};
      else
        result = cpu_reduction(h_unreduced_pixel_pairs[i][j]);
      Pair pair;
      pair.a = unique_values[i];
      pair.ax = result.a_idx % img_width;
      pair.ay = result.a_idx / img_width;
      pair.b = unique_values[j];
      pair.bx = result.b_idx % img_width;
      pair.by = result.b_idx / img_width;
      pair.distance = sqrt(result.distance);

      pairs[i][j] = pair;
      pairs[j][i] = pair.transpose();
    }
  }

  return pairs;
}
