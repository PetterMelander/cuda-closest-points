#include "hip/hip_runtime.h"
#include "../include/host_code.cuh"
#include "../include/kernels.cuh"
#include <climits>
#include <cstddef>
#include <hipcub/hipcub.hpp>
#include <tuple>
#include <utility>
#include <vector>

using std::vector;

MinResult cpu_reduction(PinnedVector<MinResult> results) {
  MinResult min_result{INT_MAX, -1, -1};
  for (MinResult &result : results) {
    if (result.distance < min_result.distance)
      min_result = result;
  }
  return min_result;
}

void sort_nonzeros(int *&d_nonzero_idxs, int *&d_nonzero_values,
                   int h_num_nonzeros) {
  // This code reinterpret casts the ints to unsigned because they are positive
  // anyway, and radix sort is faster for unsigned ints. Besides, we do not care
  // about the order, we just want identical values to be adjacent.

  int *d_idxs_buffer, *d_values_buffer;
  CUDA_CHECK(hipMalloc(&d_idxs_buffer, sizeof(int) * h_num_nonzeros));
  CUDA_CHECK(hipMalloc(&d_values_buffer, sizeof(int) * h_num_nonzeros));

  // Create a DoubleBuffer to wrap the pair of device pointers
  hipcub::DoubleBuffer<unsigned int> d_keys(
      reinterpret_cast<unsigned int *>(d_nonzero_values),
      reinterpret_cast<unsigned int *>(d_values_buffer));

  hipcub::DoubleBuffer<unsigned int> d_values(
      reinterpret_cast<unsigned int *>(d_nonzero_idxs),
      reinterpret_cast<unsigned int *>(d_idxs_buffer));

  // Determine temporary device storage requirements
  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                             d_keys, d_values, h_num_nonzeros,
                                             0, 8));

  // Allocate temporary storage
  CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));

  // Run sorting operation
  CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                             d_keys, d_values, h_num_nonzeros,
                                             0, 8));
  CUDA_CHECK(hipFree(d_temp_storage));
  bool output_in_buffer =
      (reinterpret_cast<int *>(d_keys.Current()) == d_values_buffer);
  if (output_in_buffer) {
    std::swap(d_nonzero_idxs, d_idxs_buffer);
    std::swap(d_nonzero_values, d_values_buffer);
  }
  CUDA_CHECK(hipFree(d_idxs_buffer));
  CUDA_CHECK(hipFree(d_values_buffer));
}

std::tuple<vector<int>, vector<int>, int>
encode_runs(int *d_sorted_nonzero_values, int num_nonzeros) {
  int *d_unique_values;
  CUDA_CHECK(hipMalloc(&d_unique_values, sizeof(int) * 256));
  int *d_mask_sizes;
  CUDA_CHECK(hipMalloc(&d_mask_sizes, sizeof(int) * 256));
  int *d_num_unique_values;
  CUDA_CHECK(hipMalloc(&d_num_unique_values, sizeof(int)));

  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  CUDA_CHECK(hipcub::DeviceRunLengthEncode::Encode(
      d_temp_storage, temp_storage_bytes, d_sorted_nonzero_values,
      d_unique_values, d_mask_sizes, d_num_unique_values, num_nonzeros));

  CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));

  CUDA_CHECK(hipcub::DeviceRunLengthEncode::Encode(
      d_temp_storage, temp_storage_bytes, d_sorted_nonzero_values,
      d_unique_values, d_mask_sizes, d_num_unique_values, num_nonzeros));

  CUDA_CHECK(hipFree(d_temp_storage));

  int h_num_unique_values;
  CUDA_CHECK(hipMemcpy(&h_num_unique_values, d_num_unique_values, sizeof(int),
                        hipMemcpyDeviceToHost));

  vector<int> h_unique_values(h_num_unique_values);
  vector<int> h_mask_sizes(h_num_unique_values);

  CUDA_CHECK(hipMemcpy(h_unique_values.data(), d_unique_values,
                        sizeof(int) * h_num_unique_values,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(h_mask_sizes.data(), d_mask_sizes,
                        sizeof(int) * h_num_unique_values,
                        hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(d_unique_values));
  CUDA_CHECK(hipFree(d_num_unique_values));
  CUDA_CHECK(hipFree(d_mask_sizes));

  return std::make_tuple(h_unique_values, h_mask_sizes, h_num_unique_values);
}

std::tuple<vector<int>, vector<int>, int> index_masks(const int *const h_image,
                                                      int total_pixels,
                                                      int *&d_nonzero_idxs,
                                                      int *&d_nonzero_values) {

  int *d_image;
  CUDA_CHECK(hipMalloc(&d_image, sizeof(int) * total_pixels));
  CUDA_CHECK(hipMemcpy(d_image, h_image, sizeof(int) * total_pixels,
                        hipMemcpyHostToDevice));

  // Index shapes
  int *d_num_nonzeros;
  CUDA_CHECK(hipMalloc(&d_num_nonzeros, sizeof(int)));
  CUDA_CHECK(hipMemset(d_num_nonzeros, 0, sizeof(int)));

  int num_blocks =
      num_blocks_max_occupancy(find_nonzeros, THREADS_PER_BLOCK,
                               sizeof(int) * 2 * TILE_SIZE_INDEXING, 1.5f);
  find_nonzeros<<<num_blocks, THREADS_PER_BLOCK>>>(
      d_image, total_pixels, d_nonzero_idxs, d_nonzero_values, d_num_nonzeros);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipFree(d_image));

  int h_num_nonzeros;
  CUDA_CHECK(hipMemcpy(&h_num_nonzeros, d_num_nonzeros, sizeof(int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(d_num_nonzeros));

  // Sort
  sort_nonzeros(d_nonzero_idxs, d_nonzero_values, h_num_nonzeros);

  // Encode runs
  return encode_runs(d_nonzero_values, h_num_nonzeros);
}

void launch_min_pair_thread_per_a(int num_as, int num_bs, const int img_width,
                                  int *d_as, int *d_bs, MinResult &h_result,
                                  hipStream_t stream) {

  // Kernel parallelizes over a's. Launch kernel with the larger mask as a.
  bool swapped = num_as < num_bs;
  if (swapped) {
    std::swap(num_as, num_bs);
    std::swap(d_as, d_bs);
  }

  int num_blocks =
      num_blocks_max_occupancy(min_distances_thread_per_a, THREADS_PER_BLOCK,
                               sizeof(int) * THREADS_PER_BLOCK * 3, 1.f);
  num_blocks = std::min(num_blocks,
                        (num_as + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

  MinResult *d_results;
  CUDA_CHECK(
      hipMallocAsync(&d_results, sizeof(MinResult) * num_blocks, stream));

  min_distances_thread_per_a<<<num_blocks, THREADS_PER_BLOCK, 0, stream>>>(
      d_as, d_bs, num_as, num_bs, img_width, d_results, swapped);
  CUDA_CHECK(hipGetLastError());

  final_reduction<<<1, THREADS_PER_BLOCK, 0, stream>>>(d_results, num_blocks,
                                                       d_results);
  CUDA_CHECK(hipGetLastError());

  CUDA_CHECK(hipMemcpyAsync(&h_result, d_results, sizeof(MinResult),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipFreeAsync(d_results, stream));
}

void launch_min_pair_thread_per_pair(const int num_as, const int num_bs,
                                     const int img_width, int *d_as, int *d_bs,
                                     MinResult &h_results,
                                     hipStream_t stream) {

  uint block_dim = 16;
  dim3 block_size{block_dim, block_dim};
  uint num_blocks = num_blocks_max_occupancy(
      min_distances_thread_per_pair, block_dim * block_dim,
      sizeof(MinResult) * WARP_SIZE, 1.f);
  num_blocks =
      std::min(num_blocks, (num_as * num_bs + block_dim * block_dim - 1) /
                               block_dim * block_dim);

  uint grid_dim = (uint)sqrt(num_blocks);
  dim3 grid_size{grid_dim, grid_dim};
  num_blocks = (int)grid_dim * (int)grid_dim;

  MinResult *d_results;
  CUDA_CHECK(
      hipMallocAsync(&d_results, sizeof(MinResult) * num_blocks, stream));

  int2 *d_points_a, *d_points_b;
  CUDA_CHECK(hipMallocAsync(&d_points_a, sizeof(int2) * num_as, stream));
  CUDA_CHECK(hipMallocAsync(&d_points_b, sizeof(int2) * num_bs, stream));
  make_points(d_as, d_bs, num_as, num_bs, img_width, d_points_a, d_points_b,
              stream);

  min_distances_thread_per_pair<<<grid_size, block_size, 0, stream>>>(
      d_points_a, d_points_b, num_as, num_bs, img_width, d_results);
  CUDA_CHECK(hipGetLastError());

  final_reduction<<<1, THREADS_PER_BLOCK, 0, stream>>>(d_results, num_blocks,
                                                       d_results);
  CUDA_CHECK(hipGetLastError());

  CUDA_CHECK(hipMemcpyAsync(&h_results, d_results, sizeof(MinResult),
                             hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipFreeAsync(d_points_a, stream));
  CUDA_CHECK(hipFreeAsync(d_points_b, stream));
  CUDA_CHECK(hipFreeAsync(d_results, stream));
}

void get_min_pairs(int *d_as, int *d_bs, int num_as, int num_bs, int img_width,
                   MinResult &h_result, hipStream_t stream) {
  long long num_pairs = (long long)num_as * (long long)num_bs;
  int max_mask_size = std::max(num_as, num_bs);
  if (num_pairs > (long long)INT_MAX || max_mask_size > 5000) {
  launch_min_pair_thread_per_a(num_as, num_bs, img_width, d_as, d_bs, h_result,
                               stream);
  } else {
  launch_min_pair_thread_per_pair(num_as, num_bs, img_width, d_as, d_bs,
                                  h_result, stream);
  }
}

vector<PinnedVector<MinResult>>
initial_pair_reductions(vector<int> unique_values, vector<int> mask_sizes,
                        int num_unique_values, int *d_sorted_idxs,
                        int img_width) {

  int num_mask_combinations = (num_unique_values * (num_unique_values - 1)) / 2;
  int num_streams = std::max(num_mask_combinations, 32);
  vector<hipStream_t> streams(num_streams);
  for (auto &stream : streams) {
    CUDA_CHECK(hipStreamCreate(&stream));
  }

  vector<PinnedVector<MinResult>> h_results(
      num_unique_values, PinnedVector<MinResult>(num_unique_values));

  int stream_number = 0;
  int a_offset = 0;
  for (int i = 0; i < num_unique_values; ++i) {
    int a = unique_values[i];
    int num_as = mask_sizes[i];
    int *a_ptr = d_sorted_idxs + a_offset;

    int b_offset = a_offset + num_as;
    for (int j = i + 1; j < num_unique_values; ++j) {
      int num_bs = mask_sizes[j];
      int b = unique_values[j];
      int *b_ptr = d_sorted_idxs + b_offset;

      get_min_pairs(a_ptr, b_ptr, num_as, num_bs, img_width, h_results[i][j],
                    streams[stream_number % num_streams]);

      b_offset += num_bs;
      ++stream_number;
    }
    a_offset += num_as;
  }

  for (auto &stream : streams) {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  return h_results;
}

vector<vector<Pair>> get_pairs(const int *const h_image, const int img_width,
                               const int img_height) {
  int total_pixels = img_width * img_height;

  int *d_sorted_idxs;
  int *d_sorted_values;
  CUDA_CHECK(hipMalloc(&d_sorted_idxs, sizeof(int) * total_pixels));
  CUDA_CHECK(hipMalloc(&d_sorted_values, sizeof(int) * total_pixels));

  // Index all masks
  auto [unique_values, mask_sizes, num_unique_values] =
      index_masks(h_image, total_pixels, d_sorted_idxs, d_sorted_values);
  CUDA_CHECK(hipFree(d_sorted_values));

  // Calculate (not completely reduced) pixel pairings between all masks
  vector<PinnedVector<MinResult>> h_pixel_pairs = initial_pair_reductions(
      unique_values, mask_sizes, num_unique_values, d_sorted_idxs, img_width);
  CUDA_CHECK(hipFree(d_sorted_idxs));

  // For each mask pair, do final reduction on cpu
  vector<vector<Pair>> pairs(num_unique_values,
                             vector<Pair>(num_unique_values));
  for (int i = 0; i < num_unique_values; ++i) {
    for (int j = i; j < num_unique_values; ++j) {
      MinResult result;
      if (i == j)
        result = MinResult{0, -1, -1};
      else
        result = h_pixel_pairs[i][j];
      Pair pair;
      pair.a = unique_values[i];
      pair.ax = result.a_idx % img_width;
      pair.ay = result.a_idx / img_width;
      pair.b = unique_values[j];
      pair.bx = result.b_idx % img_width;
      pair.by = result.b_idx / img_width;
      pair.distance = sqrt(result.distance);

      pairs[i][j] = pair;
      pairs[j][i] = pair.transpose();
    }
  }

  return pairs;
}
