#include "hip/hip_runtime.h"
#include <climits>
#include <hip/hip_cooperative_groups.h>
#include <cstring>
#include <random>

#define THREADS_PER_BLOCK 512
#define TILE_SIZE 2048
#define WARP_SIZE 32

// Error checking macro
#define cudaCheckErrors(msg)                                                   \
  do {                                                                         \
    hipError_t __err = hipGetLastError();                                    \
    if (__err != hipSuccess) {                                                \
      fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", msg,                  \
              hipGetErrorString(__err), __FILE__, __LINE__);                  \
      fprintf(stderr, "*** FAILED - ABORTING\n");                              \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

__device__ void flush_buffer(int *s_buffer, int &s_count, int *g_buffer,
                             int *g_count) {
  int items_to_flush = (s_count < TILE_SIZE) ? s_count : TILE_SIZE;

  __shared__ int global_base_idx;
  if (threadIdx.x == 0) {
    global_base_idx = atomicAdd(g_count, items_to_flush);
  }
  __syncthreads();

  for (int i = threadIdx.x; i < items_to_flush; i += blockDim.x) {
    g_buffer[global_base_idx + i] = s_buffer[i];
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    if (s_count > TILE_SIZE) {
      s_count = s_count - TILE_SIZE;
    } else {
      s_count = 0;
    }
  }
  __syncthreads();
}

__device__ void flush_buffer_final(int *s_buffer, int s_count, int *g_buffer,
                                   int *g_count) {
  if (s_count <= 0) {
    return;
  }

  __shared__ int global_base_idx;
  if (threadIdx.x == 0) {
    global_base_idx = atomicAdd(g_count, s_count);
  }
  __syncthreads();

  for (int i = threadIdx.x; i < s_count; i += blockDim.x) {
    g_buffer[global_base_idx + i] = s_buffer[i];
  }
}

__global__ void index_shapes(int *img_array, int dsize, int *ones,
                             int *ones_count, int *twos, int *twos_count) {
  __shared__ int s_ones[TILE_SIZE];
  __shared__ int s_twos[TILE_SIZE];
  __shared__ int s_num_ones;
  __shared__ int s_num_twos;

  if (threadIdx.x == 0) {
    s_num_ones = 0;
    s_num_twos = 0;
  }
  __syncthreads();

  int block_base_idx = blockIdx.x * blockDim.x;

  while (block_base_idx < dsize) {
    int my_idx = block_base_idx + threadIdx.x;
    int my_value = -1;
    int write_idx_ones = -1;
    int write_idx_twos = -1;

    if (my_idx < dsize) {
      my_value = img_array[my_idx];
    }

    if (my_value == 1) {
      write_idx_ones = atomicAdd(&s_num_ones, 1);
      if (write_idx_ones < TILE_SIZE) {
        s_ones[write_idx_ones] = my_idx;
      }
    } else if (my_value == 2) {
      write_idx_twos = atomicAdd(&s_num_twos, 1);
      if (write_idx_twos < TILE_SIZE) {
        s_twos[write_idx_twos] = my_idx;
      }
    }

    __syncthreads();

    bool needs_flush_ones = s_num_ones >= TILE_SIZE;
    bool needs_flush_twos = s_num_twos >= TILE_SIZE;

    if (needs_flush_ones) {
      flush_buffer(s_ones, s_num_ones, ones, ones_count);
    }
    if (needs_flush_twos) {
      flush_buffer(s_twos, s_num_twos, twos, twos_count);
    }

    if (needs_flush_ones && write_idx_ones >= TILE_SIZE) {
      s_ones[write_idx_ones - TILE_SIZE] = my_idx;
    }
    if (needs_flush_twos && write_idx_twos >= TILE_SIZE) {
      s_twos[write_idx_twos - TILE_SIZE] = my_idx;
    }

    __syncthreads();

    block_base_idx += gridDim.x * blockDim.x;
  }

  flush_buffer_final(s_ones, s_num_ones, ones, ones_count);
  flush_buffer_final(s_twos, s_num_twos, twos, twos_count);
}

struct MinResult {
  int distance;
  int one_idx;
  int two_idx;
};

__device__ __forceinline__ MinResult
warp_shuffle_reduction(MinResult min_result) {
  unsigned mask = 0xFFFFFFFFU;
  for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
    int new_distance = __shfl_down_sync(mask, min_result.distance, offset);
    int new_one_idx = __shfl_down_sync(mask, min_result.one_idx, offset);
    int new_two_idx = __shfl_down_sync(mask, min_result.two_idx, offset);
    if (new_distance < min_result.distance) {
      min_result.distance = new_distance;
      min_result.one_idx = new_one_idx;
      min_result.two_idx = new_two_idx;
    }
  }
  return min_result;
}

__global__ void min_distances(int *ones, int *twos, int num_ones, int num_twos,
                              int img_width, MinResult *block_results) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  __shared__ int s_two_idxs[THREADS_PER_BLOCK];
  __shared__ int s_x2s[THREADS_PER_BLOCK];
  __shared__ int s_y2s[THREADS_PER_BLOCK];
  int min_one_idx, x1, y1;
  int min_two_idx = -1;
  int min_distance = INT_MAX;

  if (idx < num_ones) {
    min_one_idx = ones[idx];
    x1 = min_one_idx % img_width;
    y1 = min_one_idx / img_width;
  }
  // Loop over all twos in blocks, loading into shared memory. Each thread finds its closest two.
  for (int block = 0; block < (num_twos + blockDim.x - 1) / blockDim.x;
       ++block) {
    int twos_index = block * blockDim.x + threadIdx.x;
    if (twos_index < num_twos) {
      int two = twos[twos_index];
      s_two_idxs[threadIdx.x] = two;
      s_x2s[threadIdx.x] = two % img_width;
      s_y2s[threadIdx.x] = two / img_width;
    } else
      s_two_idxs[threadIdx.x] = -1;
    __syncthreads();

    if (idx < num_ones) {
      for (int j = 0; j < THREADS_PER_BLOCK; ++j) {
        if (block * blockDim.x + j == num_twos)
          break;
        int two_idx = s_two_idxs[j];
        int x2 = s_x2s[j];
        int y2 = s_y2s[j];
        int d1 = x2 - x1;
        int d2 = y2 - y1;
        int distance = d1 * d1 + d2 * d2;
        if (distance < min_distance) {
          min_distance = distance;
          min_two_idx = two_idx;
        }
      }
    }
    __syncthreads();
  }

  // Warp shuffle reduction
  __shared__ MinResult s_min_results[WARP_SIZE];
  int lane = threadIdx.x % warpSize;
  int warpID = threadIdx.x / warpSize;
  MinResult min_result;
  min_result.distance = min_distance;
  min_result.one_idx = min_one_idx;
  min_result.two_idx = min_two_idx;

  // First warp reduction
  min_result = warp_shuffle_reduction(min_result);
  if (lane == 0)
    s_min_results[warpID] = min_result;
  __syncthreads();

  // Second warp reduction
  if (warpID == 0) {
    if (threadIdx.x < blockDim.x / warpSize)
      min_result = s_min_results[lane];
    else
      min_result.distance = INT_MAX;
    min_result = warp_shuffle_reduction(min_result);

    // Write results to global memory
    if (threadIdx.x == 0)
      block_results[blockIdx.x] = min_result;
  }
}

int main() {
  int img_size = 1024;
  int total_pixels = img_size * img_size;
  int *image = new int[total_pixels];

  // Use random device and mt19937 for reproducibility
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> dis(0.0, 1.0);

  // Assign 10% to 1, 10% to 2, rest to 0
  for (int i = 0; i < total_pixels; ++i) {
    double r = dis(gen);
    if (r < 0.1)
      image[i] = 1;
    else if (r < 0.2)
      image[i] = 2;
    else
      image[i] = 0;
  }

  int device_id;
  hipGetDevice(&device_id);
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, device_id);

  const size_t shared_mem_per_block = sizeof(int) * 2048 * 2;

  int max_active_blocks_per_sm;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks_per_sm,
                                                index_shapes, THREADS_PER_BLOCK,
                                                shared_mem_per_block);

  const int oversubscription_factor = 4;
  int num_blocks_indexing = props.multiProcessorCount *
                            max_active_blocks_per_sm * oversubscription_factor;

  int *d_image, *d_ones, *d_twos, *d_num_ones, *d_num_twos;
  int h_num_ones[1];
  int h_num_twos[1];

  hipMalloc(&d_image, total_pixels * sizeof(int));
  cudaCheckErrors("hipMalloc d_image error");
  hipMalloc(&d_ones, total_pixels * sizeof(int));
  cudaCheckErrors("hipMalloc d_ones error");
  hipMalloc(&d_twos, total_pixels * sizeof(int));
  cudaCheckErrors("hipMalloc d_twos error");
  hipMalloc(&d_num_ones, sizeof(int));
  cudaCheckErrors("hipMalloc d_num_ones error");
  hipMalloc(&d_num_twos, sizeof(int));
  cudaCheckErrors("hipMalloc d_num_twos error");

  hipMemcpy(d_image, image, total_pixels * sizeof(int),
             hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy d_image error");
  hipMemset(d_num_ones, 0, sizeof(int));
  cudaCheckErrors("hipMemset d_num_ones error");
  hipMemset(d_num_twos, 0, sizeof(int));
  cudaCheckErrors("hipMemset d_num_twos error");

  index_shapes<<<num_blocks_indexing, THREADS_PER_BLOCK>>>(
      d_image, total_pixels, d_ones, d_num_ones, d_twos, d_num_twos);
  cudaCheckErrors("index_shapes kernel launch error");
  hipMemcpy(h_num_ones, d_num_ones, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors("hipMemcpy h_num_ones error");
  hipMemcpy(h_num_twos, d_num_twos, sizeof(int), hipMemcpyDeviceToHost);
  cudaCheckErrors("hipMemcpy h_num_twos error");

  int num_blocks_reduction =
      (*h_num_ones + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  MinResult *block_results;
  hipMalloc(&block_results, sizeof(MinResult) * num_blocks_reduction);
  cudaCheckErrors("hipMalloc block_results error");
  min_distances<<<num_blocks_reduction, THREADS_PER_BLOCK>>>(
      d_ones, d_twos, *h_num_ones, *h_num_twos, img_size, block_results);
  cudaCheckErrors("min_distances kernel launch error");

  // DO THIS FINAL REDUCTION ON GPU
  int h_accum_ones[num_blocks_reduction];
  int h_accum_twos[num_blocks_reduction];
  int h_distances[num_blocks_reduction];
  // hipMemcpy(h_accum_ones, d_accum_ones, num_blocks_reduction * sizeof(int),
  //  hipMemcpyDeviceToHost);
  // cudaCheckErrors("hipMemcpy h_accum_ones error");
  // hipMemcpy(h_accum_twos, d_accum_twos, num_blocks_reduction * sizeof(int),
  //  hipMemcpyDeviceToHost);
  // cudaCheckErrors("hipMemcpy h_accum_twos error");
  // hipMemcpy(h_distances, d_accum_dist, num_blocks_reduction * sizeof(int),
  //  hipMemcpyDeviceToHost);
  // cudaCheckErrors("hipMemcpy h_distances error");

  int min_one_idx = -1;
  int min_two_idx = -1;
  int min_distance = INT_MAX;
  for (int i = 0; i < num_blocks_reduction; ++i) {
    if (h_distances[i] < min_distance) {
      min_distance = h_distances[i];
      min_one_idx = h_accum_ones[i];
      min_two_idx = h_accum_twos[i];
    }
  }

  // REMEMBER TO CONVERT FROM 1D INDEX TO 2D
  printf("Distance: %d\nIndex 1: %d\nIndex 2: %d\n", min_distance, min_one_idx,
         min_two_idx);

  delete[] image;
}

// future idea: change the distance reduction kernel to use one thread per
// combination of 1-2 indices, and use cub block reduction with operator that
// both calculates distance and selects the minimum index combination